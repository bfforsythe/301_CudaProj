#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "imageTest.h"
#include "opencv2/opencv.hpp"


void Image_Inversion_CUDA(unsigned char* Input_Image, int Height, int Width, int Channels) {
    unsigned char* Dev_Input_Image = NULL;

    hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

    
    hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

    dim3 Grid_Image(Width, Height);
    Inversion_CUDA << <Grid_Image, 1 >> > (Dev_Input_Image, Channels); 
    hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);
    hipFree(Dev_Input_Image);
    hipFree(Input_Image);
}

/*__global__ void Inversion_CUDA(unsigned char* Image, int Channels) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = (x + y * gridDim.x) * Channels;



    for (int i = 0; i < Channels; i++) {
        Image[idx + i] = 255 - Image[idx + i];
    }
} */


__global__ void GreenShift_CUDA(unsigned char* Image) {
    int x = blockIdx.x;
    int y = blockIdx.y;

    int idx = (x + y * gridDim.x);
   

    for (int i = 0; i < idx; i++) {
        Image[idx + 1] = 255 + Image[idx + 1];
    }
}