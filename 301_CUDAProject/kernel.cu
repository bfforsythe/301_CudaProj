#include "hip/hip_runtime.h"
﻿// Brandon Forsythe
// 11/18/22
// kernel.cu        (will have a better name next time)
// CUDA code in order to compliment and allow compilation of imageTest.cpp

#include "hip/hip_runtime.h" // for all cuda needs
#include "" // for copying between GPU and CPU
#include "imageTest.h" // see imageTest header. includes Image_Inversion as well as GreenShift
#include "opencv2/opencv.hpp" // for all OpenCV needs

using namespace cv; // Contrary to popular belief, this does NOT slow down the code.



// NOTE TO SELF: IT'S BGR, NOT RGB ⚠️



// ---------------------- CUDA/CV Inversion ----------------------

__global__ void Inversion_CUDA(unsigned char* Image, int Channels) { // void function to init and process image
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = (x + y * gridDim.x) * Channels;



    for (int i = 0; i < Channels; i++) {          // Image handling workhorse, what's written here will impact each pixel of the image input 
        Image[idx + i] = 255 - Image[idx + i];    // Note: OpenCV uses the BGR formula, and each channel is separated, idx + 0 is Blue
                                                                                                                    // idx + 1 is Green
                                                                                                                    // idx + 2 is Red
    }
}

void Image_Inversion_CUDA(unsigned char* Input_Image, int Height, int Width, int Channels) { // Cuda setup in order for the thing to.... actually work.
    unsigned char* Dev_Input_Image = NULL; // Dev Image by default is null, so it can be changed later.

    hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);             // Allocates as much memory as is necessary for the GPU

    
    hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice); // Copies all memory into Device (GPU)

    dim3 Grid_Image(Width, Height);                                              // Initializes image input into grid with every pixel corresponding to one square on the grid.
    Inversion_CUDA << <Grid_Image, 1 >> > (Dev_Input_Image, Channels);           // Creates workers on GPU in order to carry out above
    hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost); // Copies all memory back to Host (CPU / Computer?)

    hipFree(Dev_Input_Image);          // Frees all memory used.
    hipFree(Input_Image);
} 
// ------------------------------------------------------------------


//          Below is the Secondary test being worked on. To get compilation, please comment everything above this line, and uncomment below.
//          GreenShift also follows the basic steps as documented above, with more computation as it parses every orange pixel individually.
/*

__global__ void GreenShift(unsigned char* Image, int Channels) {
    int x = blockIdx.x;
    int y = blockIdx.y;
    int idx = (x + y * gridDim.x) * Channels;

    //for (int i = 0; i < Channels; i++) {
      //  Image[idx+2] = 255;

    for (int i = 0; i < Channels; i++) {
        if (Image[idx + 1] <= 190 && Image[idx+1] >= 100 && Image[idx+2] <= 255 && Image[idx+2] >= 210) {
            Image[idx + 1] = 1 + (Image[idx+1]) / 2;
            Image[idx] = 0;
            Image[idx + 2] = 0;
        }
    }


        // orange is (0,165,255)
    }




void GreenShift_CUDA(unsigned char* Input_Image, int Height, int Width, int Channels) {
    unsigned char* Dev_Input_Image = NULL;

    hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);


    hipMemcpy(Dev_Input_Image, Input_Image, Height * Width * Channels, hipMemcpyHostToDevice);

    dim3 Grid_Image(Width, Height);
    GreenShift << <Grid_Image, 1 >> > (Dev_Input_Image, Channels);
    hipMemcpy(Input_Image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);
    hipFree(Dev_Input_Image);
    hipFree(Input_Image);
}
*/

 