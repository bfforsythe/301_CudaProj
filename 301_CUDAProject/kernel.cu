#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "imageTest.h"

__global__ void Inversion_CUDA(unsigned char* Image, int Channels);
void Image_Inversion_CUDA(unsigned char* Input_image, int Height, int Width, int Channels) {

	unsigned char* Dev_Input_Image = NULL;

	// memAlloc

	hipMalloc((void**)&Dev_Input_Image, Height * Width * Channels);

	hipMemcpy(Dev_Input_Image, Input_image, Height * Width * Channels, hipMemcpyHostToDevice);

	dim3 Grid_Image;
	Grid_Image.x = Width;
	Grid_Image.y = Height;


	Inversion_CUDA << <Grid_Image, 255 >> > (Dev_Input_Image, Channels);


	hipMemcpy(Input_image, Dev_Input_Image, Height * Width * Channels, hipMemcpyDeviceToHost);

	hipFree(Dev_Input_Image);
}


__global__ void Inversion_CUDA(unsigned char* Image, int Channels) {

	int x = blockIdx.x;
	int y = blockIdx.y;
	int idx = (x + y * gridDim.x) * Channels;


	for (int i = 0; i < Channels; i++) {
		Image[idx + i] = 255 - Image[idx + i];
	}
}
